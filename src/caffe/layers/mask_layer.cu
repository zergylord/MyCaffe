#include "hip/hip_runtime.h"
// Copyright 2014 BVLC and contributors.

#include <algorithm>
#include <limits>
#include <vector>

#include "caffe/common.hpp"
#include "caffe/layer.hpp"
#include "caffe/syncedmem.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"
#include <stdio.h>

using std::max;

namespace caffe {


//TODO:switch to -1 since its less likely to pop up naturally
template <typename Dtype>
__global__ void MaskForward(const int n, const Dtype* in,
    const Dtype* mask, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = (mask[index] == -1.0) ? -1.0 : in[index] ;
  }
}
//takes two blobs, returns one. The second blob is the mask
//containing 1s for each element you want to keep and 0s elsewhere
template <typename Dtype>
Dtype MaskLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  if (Caffe::phase() == Caffe::TRAIN) {
    const Dtype* mask = bottom[1]->gpu_data();
    //caffe_gpu_rng_uniform(count, mask);
    // set thresholds (not used)
    // NOLINT_NEXT_LINE(whitespace/operators)
    //printf("pre mask");
    MaskForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, mask, top_data);
    CUDA_POST_KERNEL_CHECK;
    //printf("post mask");
  } else {
    caffe_gpu_copy(count, bottom_data, top_data);
  }
  return Dtype(0);
}

template <typename Dtype>
void MaskLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const bool propagate_down,
    vector<Blob<Dtype>*>* bottom) {
  CHECK(Caffe::phase() == Caffe::TRAIN);
  if (propagate_down) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
    const int count = (*bottom)[0]->count(); //should this be top?
    caffe_gpu_copy(count, top_diff, bottom_diff);//copy top grad to bottom
  }
}

INSTANTIATE_CLASS(MaskLayer);


}  // namespace caffe
