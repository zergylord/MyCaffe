#include "hip/hip_runtime.h"
#include <algorithm>
#include <limits>
#include <vector>

#include "caffe/common.hpp"
#include "caffe/layer.hpp"
#include "caffe/syncedmem.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"
#include <stdio.h>

using std::max;

namespace caffe {

template <typename Dtype>
Dtype QNetLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  int count = bottom[0]->count();
  int num = bottom[0]->num();
    caffe_gpu_copy(count,bottom[1]->gpu_data(),difference_.mutable_gpu_data());
    caffe_gpu_add_scalar(count,Dtype(1),difference_.mutable_gpu_data());//turn -1.0 -> 0.0
    caffe_gpu_scal(count,Dtype(-1),difference_.mutable_gpu_data());//turn add->sub
    caffe_gpu_cond_add(count,
            bottom[0]->gpu_data(),//addend 1
            difference_.gpu_data(),//mask
            difference_.mutable_gpu_data());//addend 2 (and the storage place)
    caffe_gpu_cond_add_scalar(count,Dtype(1),difference_.gpu_data(),difference_.mutable_gpu_data());//undo the first +1
  //Dtype loss = 1337;
  Dtype loss = caffe_cpu_asum(count,difference_.cpu_data()) / 192;//32*6, so average per action
/*//  if(loss > 100000){
      printf("hello");
      for(int i = 0;i<192;i++){
        printf("%f ",bottom[0]->cpu_data()[i]);
        if (i % 6 == 0)
            printf("\n");
      }
//  }
  if(loss > 100000){
      printf("hello");
      for(int i = 0;i<192;i++){
        printf("%f ",difference_.cpu_data()[i]);
        if (difference_.cpu_data()[i] == 0)
            printf("yay!");
        if (i % 6 == 0)
            printf("\n");
      }
  }*/
 // printf("fuck %f %d",loss,num);
  return loss;
}

template <typename Dtype>
void QNetLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const bool propagate_down, vector<Blob<Dtype>*>* bottom) {
/*  if(loss > 100000){
      printf("hello");
      for(int i = 0;i<192;i++){
        printf("%f ",difference_.cpu_data()[i]);
        if (i % 6 == 0)
            printf("\n");
      }
//  }*/
  //change to top count
  int count = (*bottom)[0]->count();
  int num = (*bottom)[0]->num();
  if (propagate_down)
    caffe_gpu_copy(count,difference_.gpu_data(),(*bottom)[0]->mutable_gpu_diff());
  else
    printf("no propagate down!");
}

INSTANTIATE_CLASS(QNetLossLayer);


}  // namespace caffe
